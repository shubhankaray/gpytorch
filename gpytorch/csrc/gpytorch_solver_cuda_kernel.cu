#include "hip/hip_runtime.h"
#include "gpytorch_solver_cuda.h"
#include <hip/hip_runtime.h>
#include <hipsolver.h>


// Parameters
const double tol = 1.e-7;
const int max_sweeps = 15;
const int sort_eig = 1;
const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;


std::tuple<at::Tensor, at::Tensor> _batch_flattened_symeig_cuda_kernel(
  at::Tensor & evals_out, at::Tensor & mat,
) {
	hipsolverHandle_t cusolver_h = NULL;
	hipStream_t stream = NULL;
	hipsolverSyevjInfo_t syevj_params = NULL;

	hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
	hipError_t cuda_stat = hipSuccess;

	const int matrix_size = mat.size(1);  // == mat.size(2)
	const int batch_size = mat.size(0);

	/* step 1: create cusolver handle, bind a stream  */
	status = hipsolverDnCreate(&cusolver_h);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	cuda_stat = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	assert(hipSuccess == cuda_stat);

	status = hipsolverSetStream(cusolver_h, stream);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	/* step 2: configuration of syevj */
	status = hipsolverDnCreateSyevjInfo(&syevj_params);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	/* default value of tolerance is machine zero */
	status = hipsolverDnXsyevjSetTolerance(syevj_params, tol);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	/* default value of max. sweeps is 100 */
	status = hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	/* sorting */
	status = hipsolverDnXsyevjSetSortEig(syevj_params, sort_eig);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	// Get data
  auto evecs_data = evecs_out.data()
  auto evals_data = evals_out.data()

	/* step 4: query working space of syevjBatched */
  int lwork = 0;
  double *work = NULL;

	status = hipsolverDnDsyevjBatched_bufferSize(
			cusolver_h,
			jobz,
			uplo,
			matrix_size,
			evec_data,
			matrix_size,
			evals_data,
			&lwork,
			syevj_params,
			batch_size
	);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

  cuda_stat = hipMalloc((void**)&d_work, sizeof(double)*lwork);
  assert(hipSuccess == cuda_stat);

  return std::make_tuple(evals_out, evecs_out);
}
